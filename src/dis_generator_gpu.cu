#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include <LHAPDF/LHAPDF.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

// Define LHAPDF globally (runs on CPU)
LHAPDF::PDF* pdf = LHAPDF::mkPDF("CT18NNLO", 0); // Load PDF

// GPU Kernel
__global__ void generate_events_GPU(double *x, double *Q2, double *weights, double *pdf_values, int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState state;
    for (int i = idx; i < N; i += stride) {
        hiprand_init(1234, i, 0, &state);
        x[i] = 0.01 + (hiprand_uniform(&state) * 0.99);  // x ∈ (0.01,1.0)
        Q2[i] = hiprand_uniform(&state) * 100.0;  // Q2 ∈ (0,100)
        
        // Interpolate f_q(x, Q2) from the precomputed table
        int x_idx = min((int)(x[i] * 100), 99); // Convert x to an index
        int Q2_idx = min((int)(Q2[i] / 1.0), 99); // Convert Q2 to an index
        double f_q = pdf_values[x_idx * 100 + Q2_idx];

        weights[i] = (2 * M_PI * 1/137.0 * 1/137.0) / (x[i] * Q2[i] * Q2[i]) * f_q;
    }
}

int main(){
    auto start = high_resolution_clock::now();

    int N_events = 10000000;
    double *x, *Q2, *weights;
    hipMallocManaged(&x, N_events * sizeof(double));
    hipMallocManaged(&Q2, N_events * sizeof(double));
    hipMallocManaged(&weights, N_events * sizeof(double));

    // Step 1: Precompute LHAPDF values on CPU
    double pdf_values[100 * 100];  // Store PDF values for interpolation
    for (int i = 0; i < 100; i++) {
        double x_val = 0.01 + i * 0.01;
        for (int j = 0; j < 100; j++) {
            double Q2_val = j * 1.0;
            pdf_values[i * 100 + j] = pdf->xfxQ(2, x_val, sqrt(Q2_val));  // Get u-quark PDF
        }
    }

    // Step 2: Copy PDF values to GPU
    double *d_pdf_values;
    hipMalloc(&d_pdf_values, 100 * 100 * sizeof(double));
    hipMemcpy(d_pdf_values, pdf_values, 100 * 100 * sizeof(double), hipMemcpyHostToDevice);

    // Step 3: Run GPU kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N_events + threadsPerBlock - 1) / threadsPerBlock;
    generate_events_GPU<<<blocksPerGrid, threadsPerBlock>>>(x, Q2, weights, d_pdf_values, N_events);
    hipDeviceSynchronize();

    // Step 4: Write results to file
    ofstream output("/home/ubuntu/DIS_EventGenerator/data/events_gpu.dat");
    for (int i = 0; i < N_events; i++){
        output << x[i] << " " << Q2[i] << " " << weights[i] << endl;
    }
    output.close();

    // Cleanup
    hipFree(x); hipFree(Q2); hipFree(weights); hipFree(d_pdf_values);

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop - start);

    cout << "Generated " << N_events << " events on GPU!" << endl;
    cout << "Time taken: " << duration.count() / 1000.0 << " seconds" << endl;

    return 0;
}

// /usr/local/cuda/bin/nvcc src/dis_generator_gpu.cu -o build/dis_generator_gpu -O3     -I/usr/local/include/LHAPDF -L/usr/local/lib -lLHAPDF     -Xcompiler -fopenmp